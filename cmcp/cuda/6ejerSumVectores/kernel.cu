
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {

    // Calculate global thread index based on the block and thread indices ----

    //INSERT KERNEL CODE HERE

    int myID = threadIdx.x + blockDim.x * blockIdx.x;

    // Use global index to determine which elements to read, add, and write ---

    //INSERT KERNEL CODE HERE
    if (myID < n)
    {
        C[myID] = A[myID] + B[myID];
    }

}

